#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH 4
#define MASK_WIDTH 3
#define MASK_RADIUS 1
//@@ Define constant memory for device kernel here
__constant__ float deviceKernel[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int bz = blockIdx.z;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;

  int Row = by*TILE_WIDTH+ty;
  int Col = bx*TILE_WIDTH+tx;
  int Hei = bz*TILE_WIDTH+tz;

  int Row_s = Row - MASK_RADIUS;
  int Col_s = Col - MASK_RADIUS;
  int Hei_s = Hei - MASK_RADIUS;
  
  __shared__ float N_ds[TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1];

  if ((Row_s >=0 && Row_s < y_size) && (Col_s>=0 && Col_s < x_size) && (Hei_s >=0 && Hei_s < z_size)){
    N_ds[tz][ty][tx] = input[Hei_s*y_size*x_size + Row_s*x_size + Col_s];
  }
  else{
   N_ds[tz][ty][tx] = 0.0; 
  }
  
  __syncthreads();
  
  float Pvalue = 0;
  if(tz < TILE_WIDTH && ty < TILE_WIDTH && tx < TILE_WIDTH){
    for(int i = 0; i < MASK_WIDTH; i++) { 
      for(int j = 0; j < MASK_WIDTH; j++) {
        for (int k = 0; k < MASK_WIDTH; k++ ) {
          Pvalue += deviceKernel[i][j][k] * N_ds[i+tz][j+ty][k+tx];  
        }
      }
    }
    __syncthreads();
    
    if( Row < y_size && Col < x_size && Hei < z_size){
      output[Hei*y_size*x_size + Row*x_size + Col] = Pvalue;
    }
  }





}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceInput, (inputLength-3) *sizeof(float));
  hipMalloc((void **) &deviceOutput, (inputLength-3) *sizeof(float));
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  hipMemcpy(deviceInput, hostInput+3, (inputLength-3)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(deviceKernel), hostKernel, kernelLength*sizeof(float), 0, hipMemcpyHostToDevice);
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimGrid(ceil((1.0*x_size)/TILE_WIDTH), ceil((1.0*y_size)/TILE_WIDTH),ceil((1.0*z_size)/TILE_WIDTH));
  dim3 dimBlock(TILE_WIDTH+MASK_WIDTH-1, TILE_WIDTH+MASK_WIDTH-1, TILE_WIDTH+MASK_WIDTH-1);
  //@@ Launch the GPU kernel here
  conv3d<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, z_size,
                       y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  hipMemcpy(hostOutput+3, deviceOutput, (inputLength-3)*sizeof(float), hipMemcpyDeviceToHost);
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
