#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  int Row = blockIdx.y*blockDim.y+threadIdx.y;
  int Col = blockIdx.x*blockDim.x+threadIdx.x;

  if ((Row < numCRows) && (Col < numCColumns)){
    float Pvalue = 0;
    for (int k=0; k< numAColumns; ++k){
      Pvalue += A[Row*numAColumns+k]* B[k*numBColumns+Col];
    }
    C[Row*numCColumns+Col] = Pvalue;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns); 
  //@@ Set numCRows and numCColumns Done
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix Done
  hostC = (float *)malloc(numCRows*numCColumns*sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here Done
  hipMalloc((void **) &deviceA, numARows*numAColumns*sizeof(float));
  hipMalloc((void **) &deviceB, numBRows*numBColumns*sizeof(float));
  hipMalloc((void **) &deviceC, numCRows*numCColumns*sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here Done
  hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here Done
  dim3 dimGrid(ceil((1.0*numCColumns)/16.0), ceil((1.0*numCRows)/16.0), 1);
  dim3 dimBlock(16, 16, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here Done
  matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows,
                               numAColumns,numBRows,
                               numBColumns,numCRows,
                               numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here Done
  hipMemcpy(hostC, deviceC, numCRows*numCColumns*sizeof(float), hipMemcpyDeviceToHost);
  
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here Done
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
