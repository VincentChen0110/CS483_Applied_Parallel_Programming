#include "hip/hip_runtime.h"
// MP 5.1 Reduction
// Given a list of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void total(float *input, float *output, int len) {
  //@@ Load a segment of the input vector into shared memory
  //@@ Traverse the reduction tree
  //@@ Write the computed sum of the block to the output vector at the
  //@@ correct index
  __shared__ float partialSum[2*BLOCK_SIZE];

  unsigned int t = threadIdx.x;
  unsigned int start = 2*blockIdx.x*blockDim.x;
  if (start+t<len){
    partialSum[t] = input[start+t];
  }
  else{
     partialSum[t] = 0;
  }
  if(start+blockDim.x+t<len){
    partialSum[blockDim.x+t] = input[start+blockDim.x+t];
  }
  else{
    partialSum[blockDim.x+t] =  0;
  }
  // partialSum[t] = input[start+t];
  // partialSum[blockDim.x+t] = input[start+blockDim.x+t];
  for(unsigned int stride = blockDim.x; stride >=1; stride >>=1){
    __syncthreads();
    if(t<stride){
      partialSum[t] += partialSum[t+stride];
    }
  }
  if(t==0)
    output[blockIdx.x] = partialSum[0];
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  numOutputElements = (numInputElements - 1) / (BLOCK_SIZE << 1) + 1;
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);
  wbLog(TRACE, "The number of output elements in the input is ",
        numOutputElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int InputSize = numInputElements*sizeof(float);
  int OutputSize = numOutputElements*sizeof(float);
  hipMalloc((void **) &deviceInput, InputSize);
  hipMalloc((void **) &deviceOutput, OutputSize);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, InputSize, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");
  //@@ Initialize the grid and block dimensions here
  dim3 DimBlock(BLOCK_SIZE,1,1);
  dim3 DimGrid(ceil(numInputElements/(BLOCK_SIZE*2.0)),1,1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, OutputSize, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  /***********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input.
   * For simplicity, we do not require that for this lab!
   ***********************************************************************/
  for (ii = 1; ii < numOutputElements; ii++) {
    hostOutput[0] += hostOutput[ii];
  }

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}
